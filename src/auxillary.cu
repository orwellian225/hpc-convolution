
#include <hip/hip_runtime.h>
#include "pnm.hpp"
#include "fmt/core.h"

#include "auxillary.cuh"

void handle_cuda_error(hipError_t error) {
    if (error == hipSuccess)
        return;
    
    fmt::println(stderr, "CUDA Error:");
    fmt::println(stderr, "\t{}", hipGetErrorString(error));
    hipDeviceReset();
    exit(EXIT_FAILURE);
}