#include <chrono>

#include <fmt/core.h>
#include <pnm.hpp>
#include <hip/hip_runtime.h>

#include "auxillary.cuh"
#include "matrix.hpp"
#include "serial.hpp"
#include "globalmem.cuh"
#include "sharedmem.cuh"

int main(int argc, char **argv) {
    fmt::println("COMS4040A High Performance Computing Assignment 1");
    fmt::println("Brendan Griffiths - 2426285");
    fmt::println("Convolution on Portable Gray Map images");
    fmt::println("{:-<80}", "-");

    if (argc != 4) {
        fmt::println(stderr, "Incorrect arguments - please specify a pgm image to load, a resultant pgm file and the convolution kernel");
        return -1;
    }

    std::string pgm_infilepath(argv[1]);
    std::string pgm_outfilepath(argv[2]);
    uint8_t selected_kernel = atoi(argv[3]) - 1;

    const std::array<std::string, 3> available_kernel_names({
        "Average", "Sharpen", "Emboss"
    });
    std::array<Matrix, 3> available_kernels({
        Matrix(5, 5, 0.04), Matrix(3, 3, -1.), Matrix(5, 5, 0.)
    });
    available_kernels[1].data[4] = 9.;
    available_kernels[2].data[0] = 1.; available_kernels[2].data[6] = 1.;
    available_kernels[2].data[18] = -1.; available_kernels[2].data[24] = -1.;

    if (selected_kernel > 2) {
        fmt::println(stderr, "Invalid kernel selected - Defaulting to average kernel");
        fmt::println(stderr, "Available Kernels:");
        for (size_t i = 0; i < 3; ++i)
            fmt::println("\t{} - {}", i + 1, available_kernel_names[i]);
        selected_kernel = 0;
    }

    pnm::pgm_image pgm = pnm::read_pgm_binary(pgm_infilepath);

    std::string serial_pgm_out_filepath = fmt::format("{}_{}_serial.pgm", pgm_outfilepath, available_kernel_names[selected_kernel]);
    std::string global_pgm_out_filepath = fmt::format("{}_{}_global.pgm", pgm_outfilepath, available_kernel_names[selected_kernel]);
    std::string shared_pgm_out_filepath = fmt::format("{}_{}_shared.pgm", pgm_outfilepath, available_kernel_names[selected_kernel]);

    const size_t num_elements = pgm.height() * pgm.width();
    const uint32_t img_width = pgm.width(), img_height = pgm.height(); 
    const size_t num_image_bytes = num_elements * sizeof(uint8_t); // every pixel is u8 grayscale so only 1 byte
    const size_t num_float_bytes = num_elements * sizeof(float);

    fmt::println("Image Properties");
    fmt::println("----------------");
    fmt::println("\t{:<32} {}", "Input Filepath:", pgm_infilepath);
    fmt::println("\t{:<32} {}", "Serial output filepath:", serial_pgm_out_filepath);
    fmt::println("\t{:<32} {}", "Global output filepath:", global_pgm_out_filepath);
    fmt::println("\t{:<32} {}", "Shared output filepath:", shared_pgm_out_filepath);
    fmt::println("\t{:<32} {} x {}", "Dimensions:", img_width, img_height);
    fmt::println("\t{:<32} {:<10}", "Image data size (bytes):", num_image_bytes);
    fmt::println("\t\t{:<24} {:<10.3f}", "(kilobytes / kB):", num_image_bytes / 1000.0);
    fmt::println("\t\t{:<24} {:<10.3f}", "(kibibytes / kiB):", num_image_bytes / 1024.0);
    fmt::println("\t{:<32} {:<10}", "Matrix data size (bytes):", num_float_bytes);
    fmt::println("\t\t{:<24} {:<10.3f}", "(kilobytes / kB):", num_float_bytes / 1000.0);
    fmt::println("\t\t{:<24} {:<10.3f}", "(kibibytes / kiB):", num_float_bytes / 1024.0);
    fmt::println("");
    fmt::println("Kernel Properties");
    fmt::println("-----------------");
    fmt::println("{} {}", "Selected Kernel:", available_kernel_names[selected_kernel]);
    available_kernels[selected_kernel].print();
    fmt::println("");

    Matrix image_matrix(pgm);
    Matrix serial_convolved_matrix(img_width, img_height, 0.);
    Matrix globalmem_convolved_matrix(img_width, img_height, 0.);
    Matrix sharedmem_convolved_matrix(img_width, img_height, 0.);

    Matrix kernel = available_kernels[selected_kernel];

    size_t block_size = 1024;
    if (img_width < 1024) {
        block_size = img_width;
    }

    const size_t grid_size = num_elements / block_size + 1;
    float serial_duration_ms = 0., globalmem_duration_ms = 0., sharedmem_duration_ms = 0.;

    auto serial_start = std::chrono::high_resolution_clock::now();
        serial::convolve(image_matrix, kernel, serial_convolved_matrix);
    auto serial_end = std::chrono::high_resolution_clock::now();
    serial_duration_ms = std::chrono::duration_cast<std::chrono::microseconds>(serial_end - serial_start).count() / 1000.;

    Matrix *d_image_matrix = Matrix::to_device(image_matrix);
    Matrix *d_kernel = Matrix::to_device(kernel);
    Matrix *d_globalmem_matrix = Matrix::to_device(globalmem_convolved_matrix);
    Matrix *d_sharedmem_matrix = Matrix::to_device(sharedmem_convolved_matrix);

    hipEvent_t globalmem_start, globalmem_end;    
    hipEventCreate(&globalmem_start);
    hipEventCreate(&globalmem_end);
    hipEventRecord(globalmem_start, 0);
        globalmem::convolve<<<grid_size, block_size>>>(d_image_matrix, d_kernel, d_globalmem_matrix);
    hipEventRecord(globalmem_end, 0);
    hipEventSynchronize(globalmem_end);
    globalmem_convolved_matrix = Matrix::to_host(d_globalmem_matrix);
    hipEventElapsedTime(&globalmem_duration_ms, globalmem_start, globalmem_end);

    // I'm getting a really weird bug here:
    // The amount of request memory is correct
    // eg. 512x512 image with 5x5 kernel asks for 2560 floats, but I can only access 2524. The 2525 float segfaults?
    // so to get around this I'm just allocating an additional 100 floats just as a security buffer to avoid the issue
    // Need to speak to William about this
    size_t row_buffer_size = sizeof(float) * block_size * kernel.height + sizeof(float) * 100; 
    size_t kernel_buffer_size = sizeof(float) * kernel.size;
    hipEvent_t sharedmem_start, sharedmem_end;    
    hipEventCreate(&sharedmem_start);
    hipEventCreate(&sharedmem_end);
    hipEventRecord(sharedmem_start, 0);
        sharedmem::convolve<<<grid_size, block_size, row_buffer_size + kernel_buffer_size>>>(d_image_matrix, d_kernel, d_sharedmem_matrix);
    hipEventRecord(sharedmem_end, 0);
    hipEventSynchronize(sharedmem_end);
    sharedmem_convolved_matrix = Matrix::to_host(d_sharedmem_matrix);
    hipEventElapsedTime(&sharedmem_duration_ms, sharedmem_start, sharedmem_end);

    bool globalmem_correct = serial_convolved_matrix.equals(globalmem_convolved_matrix, 0.001);
    bool sharedmem_correct = serial_convolved_matrix.equals(sharedmem_convolved_matrix, 0.001);

    fmt::println("{:-<178}", "-");
    fmt::println("| {:<24} | {:<19} | {:<24} | {:<23} | {:<25} | {:<24} | {:<7} | {:<7} |", 
        "Algorithm", 
        "Execution Time (ms)", 
        "Image Throughput (MiB/s)", 
        "Image Throughput (MB/s)", 
        "Matrix Throughput (MiB/s)",
        "Matrix Throughput (MB/s)",
        "Speedup",
        "Correct"
    );
    fmt::println("{:-<178}", "-");

    fmt::println("| {:<24} | {:^19.5f} | {:^24.4f} | {:^23.4f} | {:^25.4f} | {:^24.4f} | {:^7} | {:^7} |", 
        "Serial", 
        serial_duration_ms,
        (num_image_bytes / 1024. / 1024.) / (serial_duration_ms / 1000.),
        (num_image_bytes / 1000. / 1000.) / (serial_duration_ms / 1000.),
        (num_float_bytes / 1024. / 1024.) / (serial_duration_ms / 1000.),
        (num_float_bytes / 1000. / 1000.) / (serial_duration_ms / 1000.),
        "/",
        "/"
    );

    fmt::println("| {:<24} | {:^19.5f} | {:^24.4f} | {:^23.4f} | {:^25.4f} | {:^24.4f} | {:^7.4f} | {:^7} |", 
        "Global Memory CUDA", 
        globalmem_duration_ms,
        (num_image_bytes / 1024. / 1024.) / (globalmem_duration_ms / 1000.),
        (num_image_bytes / 1000. / 1000.) / (globalmem_duration_ms / 1000.),
        (num_float_bytes / 1024. / 1024.) / (globalmem_duration_ms / 1000.),
        (num_float_bytes / 1000. / 1000.) / (globalmem_duration_ms / 1000.),
        serial_duration_ms / globalmem_duration_ms,
        globalmem_correct ? "Yes" : "No"
    );
    fmt::println("| {:<24} | {:^19.5f} | {:^24.4f} | {:^23.4f} | {:^25.4f} | {:^24.4f} | {:^7.4f} | {:^7} |", 
        "Shared Memory CUDA", 
        sharedmem_duration_ms,
        (num_image_bytes / 1024. / 1024.) / (sharedmem_duration_ms / 1000.),
        (num_image_bytes / 1000. / 1000.) / (sharedmem_duration_ms / 1000.),
        (num_float_bytes / 1024. / 1024.) / (sharedmem_duration_ms / 1000.),
        (num_float_bytes / 1000. / 1000.) / (sharedmem_duration_ms / 1000.),
        serial_duration_ms / sharedmem_duration_ms,
        sharedmem_correct ? "Yes" : "No"
    );
    fmt::println("{:-<178}", "-");

    // image_matrix.print(); fmt::println("");
    // serial_convolved_matrix.print(); fmt::println("");
    // globalmem_convolved_matrix.print(); fmt::println("");
    // sharedmem_convolved_matrix.print(); fmt::println("");

    pnm::pgm_image serial_pgm = serial_convolved_matrix.to_pnm();
    pnm::pgm_image globalmem_pgm = globalmem_convolved_matrix.to_pnm();
    pnm::pgm_image sharedmem_pgm = sharedmem_convolved_matrix.to_pnm();

    pnm::write_pgm_binary(serial_pgm_out_filepath, serial_pgm);
    pnm::write_pgm_binary(global_pgm_out_filepath, globalmem_pgm);
    pnm::write_pgm_binary(shared_pgm_out_filepath, sharedmem_pgm);

    return 0;
}