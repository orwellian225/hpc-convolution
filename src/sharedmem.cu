#include "hip/hip_runtime.h"
#include "sharedmem.cuh"
#include "auxillary.cuh"

// __global__ void sharedmem::convolve(Matrix *image, Matrix *kernel, Matrix *result) {
//     int32_t image_idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (image_idx >= image->size)
//         return;

//     int32_t image_r = image_idx / image->width;
//     int32_t image_c = image_idx % image->width;

//     extern __shared__ uint8_t arena[];
//     float *shared = (float*)(arena);

//     int32_t shared_width = blockDim.x + kernel->width - 1;
//     int32_t shared_height = kernel->height;

//     for (size_t i = 0; i < shared_height * shared_width; ++i) {
//         shared[i] = 0.;
//     }

//     int32_t copy_r_start = -(int32_t)(kernel->height / 2);
//     int32_t copy_r_end = (int32_t)(kernel->height / 2);

//     for (int32_t copy_r = copy_r_start; copy_r <= copy_r_end; ++copy_r) {
//         int32_t fetch_image_idx = (image_r + copy_r) * image->width + image_c;
//         int32_t shared_idx = (copy_r + copy_r_end) * shared_width + (threadIdx.x + kernel->width / 2); // leave columns before current thread column

//         if ( -1 < copy_r + image_r && copy_r + image_r < image->height && -1 < image_c && image_c < image->width ) {
//             shared[shared_idx] = image->data[fetch_image_idx];
//         }
//     }

//     if (threadIdx.x == 0) {
//         for (int32_t copy_c = 1; copy_c <= kernel->width / 2; ++copy_c) {
//             for (int32_t copy_r = copy_r_start; copy_r <= copy_r_end; ++copy_r) {
//                 int32_t fetch_image_idx = (image_r + copy_r) * image->width + image_c - copy_c;
//                 int32_t shared_idx = (copy_r + copy_r_end) * shared_width - copy_c + (kernel->width / 2); // always first columns in array

//                 if ( -1 < copy_r + image_r && copy_r + image_r < image->height && -1 < image_c - copy_c && image_c - copy_c < image->width ) {
//                     shared[shared_idx] = image->data[fetch_image_idx];
//                 }
//             }
//         }
//     }

//     if (threadIdx.x == image->width - 1) {
//         for (int32_t copy_c = 1; copy_c <= kernel->width / 2; ++copy_c) {
//             for (int32_t copy_r = copy_r_start; copy_r <= copy_r_end; ++copy_r) {
//                 int32_t fetch_image_idx = (image_r + copy_r) * image->width + image_c + copy_c;
//                 int32_t shared_idx = (copy_r + copy_r_end) * shared_width + image_c + copy_c; // always last columns in array

//                 if ( -1 < copy_r + image_r && copy_r + image_r < image->height && -1 < image_c + copy_c && image_c + copy_c < image->width ) {
//                     shared[shared_idx] = image->data[fetch_image_idx];
//                 }
//             }
//         }
//     }

//     __syncthreads();

//     // Get the index in shared memory of our current thread
//     int32_t shared_idx = shared_width * (shared_height / 2) + threadIdx.x + kernel->width / 2;
//     int32_t shared_r = shared_idx / shared_width;
//     int32_t shared_c = shared_idx % shared_width;
//     int32_t kernel_r_start = -(int32_t)( kernel->height / 2 ), kernel_r_end = (int32_t)( kernel->height / 2 );
//     int32_t kernel_c_start = -(int32_t)( kernel->width / 2 ), kernel_c_end = (int32_t)( kernel->width / 2 );

//     float sum = 0.;
//     for (int32_t kernel_r = kernel_r_start; kernel_r <= kernel_r_end; ++kernel_r) {
//         for (int32_t kernel_c = kernel_c_start; kernel_c <= kernel_c_end; ++kernel_c) {
//             int32_t kernel_shared_r = shared_r + kernel_r;
//             int32_t kernel_shared_c = shared_c + kernel_c;

//             int32_t kernel_idx = (kernel_r + kernel->height / 2) * kernel->height + (kernel_c + kernel->width / 2);
//             int32_t kernel_shared_idx = kernel_shared_r * shared_width + kernel_shared_c;

//             sum += shared[kernel_shared_idx] * kernel->data[kernel_idx];
//         }
//     }

//     result->data[image_idx] = sum;
// }

__global__ void sharedmem::convolve(Matrix *image, Matrix *kernel, Matrix *result) {
    size_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ uint8_t arena[];
    float *kernel_buffer = (float*)(arena);
    float *row_buffer = (float*)(arena) + sizeof(float) * kernel->size;

    if (thread_idx < kernel->size) {
        kernel_buffer[thread_idx] = kernel->data[thread_idx];
    }

    __syncthreads();

    if (thread_idx >= image->size)
        return;

    size_t r = thread_idx / image->width;
    size_t c = thread_idx % image->width;

    float sum = 0.;
    for (int32_t kernel_r = -(int32_t)(kernel->height / 2); kernel_r <= (int32_t)(kernel->height / 2); ++kernel_r) {
        for (int32_t kernel_c = -(int32_t)(kernel->width / 2); kernel_c <= (int32_t)(kernel->width / 2); ++kernel_c) {

            int32_t kernel_image_r = r + kernel_r;
            int32_t kernel_image_c = c + kernel_c;

            int32_t kernel_idx = (kernel_r + kernel->height / 2) * kernel->height + (kernel_c + kernel->width / 2);
            int32_t kernel_image_idx = kernel_image_r * image->width + kernel_image_c;

            if ( -1 < kernel_image_r && kernel_image_r < image->height && -1 < kernel_image_c && kernel_image_c < image->width )
                sum += image->data[kernel_image_idx] * kernel->data[kernel_idx];
        }
    }

    result->data[thread_idx] = sum;
}